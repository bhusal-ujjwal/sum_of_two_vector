
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <chrono>

#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

void sumVectorsCPU(int *a, int *b, int *c, int size) {
    for (int i = 0; i < size; ++i) {
        c[i] = a[i] + b[i];
    }
}

__global__ void sumVectorsGPUManual(int *a, int *b, int *c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

__global__ void sumVectorsGPUAuto(int *a, int *b, int *c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

#define vectorDim 1000

int main() {
    for (int size = 100000; size < 1000000; size += 10000) {
        int *dataA = new int[size];
        int *dataB = new int[size];
        int *dataC = new int[size];

        // Fill dataA and dataB with values
        for (int i = 0; i < size; ++i) {
            dataA[i] = i + 1;
            dataB[i] = i + 2;
        }

        // CPU version
        auto startCPU = std::chrono::steady_clock::now();
        sumVectorsCPU(dataA, dataB, dataC, size);
        auto endCPU = std::chrono::steady_clock::now();
        auto timeCPU = std::chrono::duration_cast<std::chrono::microseconds>(endCPU - startCPU).count();

        // GPU version with manual memory management
        int *d_dataA, *d_dataB, *d_dataC;
        hipMalloc((void**)&d_dataA, size * sizeof(int));
        hipMalloc((void**)&d_dataB, size * sizeof(int));
        hipMalloc((void**)&d_dataC, size * sizeof(int));

        auto startGPUManual = std::chrono::steady_clock::now();
        hipMemcpy(d_dataA, dataA, size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_dataB, dataB, size * sizeof(int), hipMemcpyHostToDevice);

        int threadsPerBlock = 256;
        int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
        sumVectorsGPUManual<<<blocksPerGrid, threadsPerBlock>>>(d_dataA, d_dataB, d_dataC, size);

        hipMemcpy(dataC, d_dataC, size * sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        auto endGPUManual = std::chrono::steady_clock::now();
        auto timeGPUManual = std::chrono::duration_cast<std::chrono::microseconds>(endGPUManual - startGPUManual).count();

        // Free GPU memory
        hipFree(d_dataA);
        hipFree(d_dataB);
        hipFree(d_dataC);

        // GPU version with automatic memory management (Unified Memory)
        int *d_dataA_auto, *d_dataB_auto, *d_dataC_auto;
        hipMallocManaged(&d_dataA_auto, size * sizeof(int));
        hipMallocManaged(&d_dataB_auto, size * sizeof(int));
        hipMallocManaged(&d_dataC_auto, size * sizeof(int));

        for (int i = 0; i < size; ++i) {
            d_dataA_auto[i] = dataA[i];
            d_dataB_auto[i] = dataB[i];
        }

        auto startGPUAuto = std::chrono::steady_clock::now();
        int blocksPerGridAuto = (size + threadsPerBlock - 1) / threadsPerBlock;
        sumVectorsGPUAuto<<<blocksPerGridAuto, threadsPerBlock>>>(d_dataA_auto, d_dataB_auto, d_dataC_auto, size);
        hipDeviceSynchronize();
        auto endGPUAuto = std::chrono::steady_clock::now();
        auto timeGPUAuto = std::chrono::duration_cast<std::chrono::microseconds>(endGPUAuto - startGPUAuto).count();

        // Free GPU memory (Unified Memory)
        hipFree(d_dataA_auto);
        hipFree(d_dataB_auto);
        hipFree(d_dataC_auto);

        // Print results
        std::cout << size << "," << timeCPU << "," << timeGPUManual << "," << timeGPUAuto << std::endl;

        delete[] dataA;
        delete[] dataB;
        delete[] dataC;
    }

    return 0;
}